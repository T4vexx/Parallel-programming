#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define A_X 0.0
#define B_X 1.5
#define A_Y 0.0
#define B_Y 1.5
#define THREADS_PER_BLOCK 512

__device__ double f(double x, double y) {
    return sin(x * x + y * y);
}

__global__ void integralKernel(double* result, int nx, int ny, double delta_x, double delta_y) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    double local_sum = 0.0;
    
    // Cada thread processa múltiplos pontos
    for(int i = tid; i <= nx; i += total_threads) {
        double x_i = A_X + i * delta_x;
        
        for(int j = 0; j <= ny; j++) {
            double y_j = A_Y + j * delta_y;
            double weight = 1.0;
            
            if((i == 0 || i == nx) && (j == 0 || j == ny)) {
                weight = 0.25; // Canto
            }
            else if(i == 0 || i == nx || j == 0 || j == ny) {
                weight = 0.5; // Borda
            }
            
            local_sum += weight * f(x_i, y_j);
        }
    }
    
    __shared__ double shared_sum[THREADS_PER_BLOCK];
    shared_sum[threadIdx.x] = local_sum;
    __syncthreads();
    
    for(int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if(threadIdx.x < stride) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    if(threadIdx.x == 0) {
        atomicAdd(result, shared_sum[0] * delta_x * delta_y);
    }
}

int main(int argc, char *argv[]) {
    if(argc != 4) {
        fprintf(stderr, "Uso: %s <num_blocks> <nx> <ny>\n", argv[0]);
        return 1;
    }
    
    int num_blocks = atoi(argv[1]);
    int nx = atoi(argv[2]);
    int ny = atoi(argv[3]);
    
    double *d_result, h_result = 0.0;
    const double delta_x = (B_X - A_X) / nx;
    const double delta_y = (B_Y - A_Y) / ny;
    
    hipMalloc(&d_result, sizeof(double));
    hipMemcpy(d_result, &h_result, sizeof(double), hipMemcpyHostToDevice);
    
    integralKernel<<<num_blocks, THREADS_PER_BLOCK>>>(d_result, nx, ny, delta_x, delta_y);
    
    hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_result);
    
    printf("Blocos: %d, Nx: %d, Ny: %d\n", num_blocks, nx, ny);
    printf("Valor aproximado da integral: %.10lf\n", h_result);
    
    return 0;
}